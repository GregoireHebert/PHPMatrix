
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <sys/time.h>

extern "C" {

void _cublas_mul(float* a, float* b, float* c, int r1, int c1, int r2, int c2) {

    //デバイス(GPU)側用 　
    float *devA,*devB,*devC;

    // 行列演算 C=αAB+βC のパラメータ
    float alpha = 1.0f;
    float beta = 0.0f;

    size_t memSzA = r1 * c1 * sizeof(float);
    size_t memSzB = r2 * c2 * sizeof(float);
    size_t memSzC = r1 * c2 * sizeof(float);

    //デバイス側メモリ確保
    hipMalloc((void **)&devA, memSzA);
    hipMalloc((void **)&devB, memSzB);
    hipMalloc((void **)&devC, memSzC);

    //ホスト → デバイス memcpy
    hipblasSetVector(r1 * c1, sizeof(float), a, 1, devA, 1);
    hipblasSetVector(r2 * c2, sizeof(float), b, 1, devB, 1);

    // デバイス側ハンドル作成
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 行列の積 演算
    // CuBLAS は列志向であることに注意する！！
    // https://stackoverflow.com/questions/13782012/how-to-transpose-a-matrix-in-cuda-cublas
    hipblasSgemm(
            handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            c2,
            r1,
            c1,
            &alpha,
            devB,
            c2,
            devA,
            c1,
            &beta,
            devC,
            c2
    );

    // デバイス側ハンドル破棄
    hipblasDestroy(handle);

    // ホスト ← デバイス memcpy (計算結果取得)
    hipblasGetVector(r1 * c2, sizeof(float), devC, 1, c, 1);

    // デバイス側メモリ解放
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

}

}
